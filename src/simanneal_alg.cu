#include "hip/hip_runtime.h"
// @file:     simanneal_alg.cu
// @author:   Samuel
// @created:  2021.07.30
// @license:  Apache License 2.0
//
// @desc:     Implementation of SimAnneal's CUDA algorithms

#include <hip/hip_runtime.h>
#include <hipblas.h>  // TODO might not use this one
#include <hiprand/hiprand_kernel.h>
#include "constants.h"
#include "phys_model.cu"

// GLOBAL VARS

__device__ hipblasHandle_t *cb_hdl;
__device__ int n_dbs, anneal_cycles, hop_attempt_factor;
__device__ float alpha, kT_start, kT_min, v_freeze_thresh, v_freeze_step, muzm, mupz;
__device__ float *v_ij, *v_ext;

// MACROS

/**
 * 0-based index for cuBLAS.
 */
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

/**
 * CUDA error checking.
 */
#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/**
 * Only include code if debug build.
 */
#ifndef NDEBUG
#define DEBUG_RUN(code) code
#else
#define DEBUG_RUN(code)
#endif


/**
 * Print 1D float array content in device.
 */
#define print1DArrayFloat(name, arr, size) \
  printf("%s=[", name); \
  for (int i=0; i<size; i++) { \
    printf("%f", arr[i]); \
    if (i!=size-1) \
      printf(", "); \
  } \
  printf("]\n");

/**
 * Print 1D int array content in device.
 */
#define print1DArrayInt(name, arr, size) \
  printf("%s=[", name); \
  for (int i=0; i<size; i++) { \
    printf("%i", arr[i]); \
    if (i!=size-1) \
      printf(", "); \
  } \
  printf("]\n");

/**
 * Print 2D float square array content in device in cublas indices.
 */
#define print2DArrayFloat(name, arr, size) \
  printf("%s=\n[", name); \
  for (int i=0; i<size; i++) {\
    if (i == 0) {\
      printf("[");\
    } else {\
      printf(" [");\
    }\
    for (int j=0; j<size; j++) {\
      printf("%f", arr[IDX2C(i,j,size)]);\
      if (j != size-1) \
        printf(", "); \
    }\
    if (i != size-1) {\
      printf("],\n"); \
    }\
  }\
  printf("]]\n");

// FUNCTIONS

/**
 * Generate a random integer in range [0, cap) and write to output pointer.
 * Note that cuRAND generates a float in range (0,1].
 * @param curand_state the cuRAND handle.
 * @param cap max int (exclusive).
 * @param output pointer to write generated int to.
 */
__device__ int randInt(hiprandState *curand_state, int cap)
{
  return static_cast<int>(ceilf(cap * hiprand_uniform(curand_state))) - 1;
}

/**
 * Return a probabilistic truth state.
 * Note that cuRAND generates a float in range (0,1].
 * @param curand_state the cuRAND handle.
 * @param prob the probability of returning true.
 * @param truth_state truth state to return.
 */
__device__ void evalProb(hiprandState *curand_state, float prob, bool *result)
{
  float rand = hiprand_uniform(curand_state);
  *result = rand < prob;
  //return hiprand_uniform(curand_state) < prob;
}

/**
 * Randomize initial population
 */
template<typename TCharge>
__device__ void randomizeChargeStates(TCharge *n, int n_dbs, hiprandState *curand_state)
{
  int t_id = threadIdx.x + (blockIdx.x * blockDim.x);
  if (t_id >= n_dbs) {
    // skip threads that have nothing to do
    return;
  }
  int stride = blockDim.x * gridDim.x;

  for (int i=t_id; i<n_dbs; i+=stride) {
    n[i] = randInt(curand_state, 3) - 1;
  }
}

/**
 * Generate a suggested population delta for the given physical conditions.
 */
template<typename TCharge, typename TFloat>
__device__ void genPopDelta(TCharge *n, TCharge *dn, bool *changed, TFloat *v_local, TFloat *v_freeze,
                            TFloat *kT, TFloat muzm, TFloat mupz, int n_dbs,
                            hiprandState *curand_state)
{
  int t_id = threadIdx.x + (blockIdx.x * blockDim.x);
  int stride = blockDim.x * gridDim.x;

  if (t_id == 0) {
    *changed = false;
  }
  __syncthreads();

  float prob;
  float x, x_zm, x_pz;
  int change_dir;
  bool accept;

  for (int i=t_id; i<n_dbs; i+=stride) {
    // new less branch attempt
    // bool start_from_db0 = n[i] == 0;
    // x_zm = v_local[i] + muzm;
    // x_pz = v_local[i] + mupz;
    // bool from_db0_closer_to_zm = fabs(x_zm) < fabs(x_pz);
    // bool start_from_dbm = n[i] == -1;
    // x = *v_freeze + start_from_db0 * (!from_db0_closer_to_zm * -1 * x_pz + from_db0_closer_to_zm * x_zm)
    //     + !start_from_db0 * (n[i] * (start_from_dbm * x_zm + !start_from_dbm * x_pz));
    // change_dir = start_from_db0 * (1 - 2 * from_db0_closer_to_zm) + !start_from_db0 * (-1 + start_from_dbm * 2);
    // Above is an attempt at reducing branching
    // Below is the original code
    if (n[i] == -1) {
      // probability from DB- to DB0
      x = - (v_local[i] + muzm) + *v_freeze;
      change_dir = 1;
    } else if (n[i] == 1) {
      // probability from DB+ to DB0
      x = v_local[i] + mupz + *v_freeze;
      change_dir = -1;
    } else {
      if (fabs(v_local[i] + muzm) < fabs(v_local[i] + mupz)) {
        // closer to DB(0/-) transition level, probability from DB0 to DB-
        x = v_local[i] + muzm + *v_freeze;
        change_dir = -1;
      } else {
        // closer to DB(+/0) transition level, probability from DB0 to DB+
        x = - (v_local[i] + mupz) + *v_freeze;
        change_dir = 1;
      }
    }
    prob = 1. / (1 + (exp(x / *kT)));

    evalProb(curand_state, prob, &accept);
    // new less branch attempt
    dn[i] = change_dir * accept;
    *changed = accept;
    // Above is an attempt to get rid of the branching by implicitly casting bool to int
    // if (accept) {
    //   dn[i] = change_dir;
    //   *changed = true;
    // } else {
    //   dn[i] = 0;
    // }
  }
}

/**
 * Choose sites to hop from (charged) and to (neutral).
 * @param dbm_occ DB- occupation indices.
 * @param db0_occ DB0 occupation indices.
 * @param dbp_occ DB+ occupation indices.
 * @param dbm_count count of DBs in -ve charge state.
 * @param db0_count count of DBs in neutral charge state.
 * @param dbp_count count of DBs in +ve charge state.
 * @param from_state charge state of the "from" DB.
 * @param from_occ_ind index of the "from" DB on the corresponding occ array.
 * @param to_occ_ind index of the "to" DB on the neutral occ array.
 * @param from_db_ind index of the DB site that the charge is hopping from.
 * @param to_db_ind index of the DB site that the charge is hopping to.
 * @param curand_state cuRAND state pointer for RNG.
 */
__device__ void chooseHopIndices(int *dbm_occ, int *db0_occ, int *dbp_occ,
                                 int dbm_count, int db0_count, int dbp_count,
                                 int *from_state, int *from_occ_ind,
                                 int *to_occ_ind, int *from_db_ind,
                                 int *to_db_ind, hiprandState *curand_state)
{
  *from_occ_ind = randInt(curand_state, dbm_count + dbp_count);

  // new less branch attempt
  bool choose_dbp = *from_occ_ind >= dbm_count;
  *from_state = 2 * choose_dbp - 1;  // equiv to choose_dbp ? 1 : -1;
  *from_occ_ind -= dbm_count * choose_dbp;
  *from_db_ind = choose_dbp ? dbp_occ[*from_occ_ind] : dbm_occ[*from_occ_ind];
  // Above is an attempt to minimize branching
  // Not sure if it helps more than it hurts though since there's still a condition
  // if (*from_occ_ind < dbm_count) {
  //   *from_state = -1;
  //   *from_db_ind = dbm_occ[*from_occ_ind];
  // } else {
  //   *from_state = 1;
  //   *from_occ_ind -= dbm_count;
  //   *from_db_ind = dbp_occ[*from_occ_ind];
  // }
  *to_occ_ind = randInt(curand_state, db0_count);
  *to_db_ind = db0_occ[*to_occ_ind];
}

/**
 * Calculate the energy delta for a hop operation.
 * @param n the charge states.
 * @param n_dbs count of DBs.
 * @param from_db_ind hopping from DB at this index.
 * @param to_db_ind hopping to DB at this index.
 * @param v_local local potential at DB sites.
 * @param v_ij pre-computed coulombic repulsion.
 * @param E_delta return change in energy resulting from this hop.
 */
template <typename TCharge, typename TFloat>
__device__ void calcHopEnergyDelta(TCharge *n, int n_dbs, int *from_state,
                                   int *from_db_ind, int *to_db_ind,
                                   TFloat *v_local, TFloat *v_ij, TFloat *E_delta)
{
  int t_id = blockIdx.x * blockDim.x + threadIdx.x;
  int dn_i = - *from_state;
  int dn_j = *from_state;
  E_delta[t_id] = -v_local[*from_db_ind] * dn_i - v_local[*to_db_ind] * dn_j - v_ij[IDX2C(*from_db_ind, *to_db_ind, n_dbs)];
}

/**
 * Return whether a hop should be accepted at this stage of annealing.
 * @param E_delta change in energy resulting from this hop.
 * @param kT current annealing temperature.
 * @param accept return boolean indicating whether this is accepted.
 * @param curand_state cuRAND handle.
 */
template <typename TFloat>
__device__ void acceptHop(TFloat *E_delta, TFloat *kT, bool *accept, hiprandState *curand_state)
{
  int t_id = blockIdx.x * blockDim.x + threadIdx.x;

  TFloat prob = fminf(1.0, exp(-(E_delta[t_id]) / (*kT)));
  evalProb(curand_state, prob, accept);

  // if (E_delta[t_id] < 0) {
  //   *accept = true;
  // } else {
  //   TFloat prob = exp(-(E_delta[t_id]) / (*kT));
  //   //*accept = evalProb(curand_state, prob);
  //   evalProb(curand_state, prob, accept);
  // }
}

/**
 * Perform the specified hop and update relevant parameters.
 * @param n charge configuration.
 * @param n_dbs DB count.
 * @param from_db_ind originate DB site.
 * @param to_db_ind destination DB site.
 * @param E_sys system energy to be updated.
 * @param E_delta previously calculated change in system energy from the hop.
 * @param v_local local potentials to be updated.
 * @param v_ij precomputed coulombic repulsion.
 */
template <typename TCharge, typename TFloat>
__device__ void performHopUpdates(TCharge *n, int n_dbs, int *from_state,
                                  int *from_db_ind, int *to_db_ind,
                                  TFloat *E_sys, TFloat *E_delta,
                                  TFloat *v_local, TFloat *v_ij)
{
  int t_id = blockIdx.x * blockDim.x + threadIdx.x;

  n[*from_db_ind] += - *from_state;
  n[*to_db_ind] += *from_state;
  E_sys[t_id] += E_delta[t_id];
  for (int i=0; i<n_dbs; i++) {
    v_local[i] = v_local[i] - ((- *from_state) * v_ij[IDX2C(i, *from_db_ind, n_dbs)] + (*from_state) * v_ij[IDX2C(i, *to_db_ind, n_dbs)]);
  }
}

/**
 * Run the main SimAnneal algorithm on the GPU.
 * 
 * @param stream_id CUDA stream ID.
 * @param n_out charge states output.
 */
__global__ void runAnneal(int stream_id, int *n_out)
{
  int t_id = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  /*
  printf("blockIdx.x=%d, blockDim.x=%d, threadIdx.x=%d\n", blockIdx.x, blockDim.x, threadIdx.x);
  printf("t_id=%d\n", t_id);
  */

  // RNG
  hiprandState curand_state;
  hiprand_init((unsigned long long)clock() + t_id + stream_id, 0, 0, &curand_state);

  // best charge state tracking
  // TODO: it might not be worth it to keep track of best config and best E because
  // the lowest energy might not be metastable, additional checks take time
  //float *best_n;      // current best charge configuration
  //float best_E;       // current best system energy

  // copy arrays over from global memory
  __shared__ float *v_ij_s, *v_ext_s;
  if (t_id == 0) {
    v_ij_s = new float[n_dbs*n_dbs];
    v_ext_s = new float[n_dbs];
    for (int i=0; i<n_dbs*n_dbs; i++) {
      v_ij_s[i] = v_ij[i];
    }
    for (int i=0; i<n_dbs; i++) {
      v_ext_s[i] = v_ext_s[i];
    }
  }

  // charge state, population, and env tracking
  __shared__ float *n, *dn;             // current/delta DB site charge state (float due to linalg)
  __shared__ bool *pop_changed;         // whether population is updated
  __shared__ float *E_sys, *E_delta;
  __shared__ float *v_local;            // local potential at each site
  if (t_id == 0) {
    n = new float[n_dbs];
    dn = new float[n_dbs];
    pop_changed = new bool;
    E_sys = new float;
    E_delta = new float;
    v_local = new float[n_dbs];
    for (int i=0; i<n_dbs; i++) {
      n[i] = 0.;
      dn[i] = 0.;
      v_local[i] = 0.;
    }
  }

  // DB-, DB0, DB+ sites tracking
  int dbm_occ_count, db0_occ_count, dbp_occ_count;
  int *dbm_occ, *db0_occ, *dbp_occ;  // DB site indices currently at -ve/neutral/+ve charge state
  dbm_occ = new int[n_dbs];
  db0_occ = new int[n_dbs];
  dbp_occ = new int[n_dbs];

  // hop tracking
  int hop_attempts, max_hop_attempts;
  __shared__ float *E_sys_hop, *E_delta_hop;
  //float *E_sys_hop, *E_delta_hop;
  float *v_local_l;
  float *n_l;
  int *from_state, *from_occ_ind, *to_occ_ind, *from_db_ind, *to_db_ind;
  bool *accept_hop;
  v_local_l = new float[n_dbs];
  n_l = new float[n_dbs];
  from_state = new int;
  from_occ_ind = new int;
  to_occ_ind = new int;
  from_db_ind = new int;
  to_db_ind = new int;
  accept_hop = new bool;
  if (t_id == 0) {
    E_sys_hop = new float[blockDim.x];
    E_delta_hop = new float[blockDim.x];
  }

  // best hop thread selection
  __shared__ int best_t_id;
  __shared__ float E_best;

  // annealing cycle tracking
  __shared__ int cycle; // current anneal cycle
  __shared__ float *v_freeze, *kT;
  if (t_id == 0) {
    cycle = 0; // current anneal cycle
    v_freeze = new float;
    kT = new float;
    *v_freeze = 0.;
    *kT = kT_start; // TODO reenable
  }

  // temp vars
  __shared__ float *temp_scalar;
  __shared__ float *temp_scalar_1;
  __shared__ float *temp_vec_ndbs_0;
  __shared__ float *temp_vec_ndbs_1;
  if (t_id == 0) {
    temp_scalar = new float;
    temp_scalar_1 = new float;
    temp_vec_ndbs_0 = new float[n_dbs];
    temp_vec_ndbs_1 = new float[n_dbs];
  }

  // randomize initial charge state and find energy
  __syncthreads();
  randomizeChargeStates(n, n_dbs, &curand_state);
  __syncthreads();
  calcSystemEnergy(n, n_dbs, v_ij_s, v_ext_s, E_sys, temp_scalar, temp_vec_ndbs_0, temp_vec_ndbs_1);
  calcLocalPotentials(n, n_dbs, v_ij_s, v_ext_s, v_local, temp_vec_ndbs_0);
  __syncthreads();
  DEBUG_RUN(
    if (t_id == 0) {
      print1DArrayFloat("randomized n", n, n_dbs);
      printf("E_sys=%.3e\n", *E_sys);
      print1DArrayFloat("v_local", v_local, n_dbs);
    }
  )

  // run algorithm
  if (t_id == 0) {
    printf("***** Begin SimAnneal Algorithm Stream %d *****\n", stream_id);
  }
  while (cycle < anneal_cycles) {
    // Population update
    genPopDelta(n, dn, pop_changed, v_local, v_freeze, kT, muzm, mupz, n_dbs, &curand_state);
    __syncthreads();
    DEBUG_RUN(
      if (t_id == 0) {
        print1DArrayFloat("dn", dn, n_dbs); // TODO: remove
      }
    )
    if (pop_changed) {
      vvAdd(n, dn, n, n_dbs); // update the charge list
      __syncthreads();
      popChangeDeltaUpdates(dn, n_dbs, v_ij_s, v_local, E_sys, temp_scalar, temp_vec_ndbs_0, temp_vec_ndbs_1);
      __syncthreads();
      DEBUG_RUN(
        if (t_id == 0) {
          print1DArrayFloat("new n", n, n_dbs); // TODO: remove
          printf("E_sys=%.3e\n", *E_sys); // TODO: remove
        }
      )
      DEBUG_RUN(
        calcSystemEnergy(n, n_dbs, v_ij_s, v_ext_s, temp_scalar, temp_scalar_1, temp_vec_ndbs_0, temp_vec_ndbs_1);
        if (t_id == 0) {
          printf("E_sys_calc=%.3e\n", *temp_scalar);
          print1DArrayFloat("v_local", v_local, n_dbs); // TODO: remove
        }
      )

      // update occupation lists
      int dbm_ind=0, db0_ind=0, dbp_ind=0;
      for (int db_ind=0; db_ind<n_dbs; db_ind++) {
        if (n[db_ind] == -1) {
          dbm_occ[dbm_ind++] = db_ind;
        } else if (n[db_ind] == 0) {
          db0_occ[db0_ind++] = db_ind;
        } else {
          dbp_occ[dbp_ind++] = db_ind;
        }
      }
      dbm_occ_count = dbm_ind;
      db0_occ_count = db0_ind;
      dbp_occ_count = dbp_ind;
      DEBUG_RUN(
        if (t_id == 0) {
          printf("DB-: %d, DB0: %d, DB+: %d\n", dbm_occ_count, db0_occ_count, dbp_occ_count);
          print1DArrayInt("dbm_occ", dbm_occ, n_dbs);
          print1DArrayInt("db0_occ", db0_occ, n_dbs);
          print1DArrayInt("dbp_occ", dbp_occ, n_dbs);
        }
      )
    }

    // calculate how many hops should be attempted
    __syncthreads();
    hop_attempts = 0;
    max_hop_attempts = 0;
    if (dbm_occ_count + dbp_occ_count < n_dbs && db0_occ_count < n_dbs) {
      max_hop_attempts = max(dbm_occ_count+dbp_occ_count, db0_occ_count);
      max_hop_attempts *= hop_attempt_factor;
    }

    // copy v_local and n to local thread
    __syncthreads();
    E_sys_hop[t_id] = *E_sys;
    for (int i=0; i<n_dbs; i++) {
      v_local_l[i] = v_local[i];
      n_l[i] = n[i];
    }

    // Hopping
    __syncthreads();
    DEBUG_RUN(
      if (t_id == 0) {
        print1DArrayFloat("E_sys_hop before", E_sys_hop, blockDim.x);
      }
    )
    while (hop_attempts < max_hop_attempts) {
      chooseHopIndices(dbm_occ, db0_occ, dbp_occ, dbm_occ_count, db0_occ_count,
                       dbp_occ_count, from_state, from_occ_ind, to_occ_ind,
                       from_db_ind, to_db_ind, &curand_state);
      __syncthreads();
      DEBUG_RUN(
        // TODO: loop through all threads to print their corresponding hops instead
        if (t_id == 0) {
          printf("Chose to hop from %d to %d (from state is %d)\n", *from_db_ind, *to_db_ind, *from_state);
        }
      )
      calcHopEnergyDelta(n_l, n_dbs, from_state, from_db_ind, to_db_ind, v_local_l, v_ij_s, E_delta_hop);
      __syncthreads();
      acceptHop(E_delta_hop, kT, accept_hop, &curand_state);
      __syncthreads();
      if (*accept_hop) {
        DEBUG_RUN(
          if (t_id == 0) {
            printf("ACCEPTED\n");
          }
        )
        // hop accepted, update telemetry
        performHopUpdates(n_l, n_dbs, from_state, from_db_ind, to_db_ind, E_sys_hop, E_delta_hop, v_local_l, v_ij_s);
        //__syncthreads();
        DEBUG_RUN(
          if (t_id == 0) {
            print1DArrayFloat("n after hop", n_l, n_dbs);
          }
        )
        if (*from_state == -1) {
          dbm_occ[*from_occ_ind] = *to_db_ind;
        } else {
          dbp_occ[*from_occ_ind] = *to_db_ind;
        }
        db0_occ[*to_occ_ind] = *from_db_ind;
      }

      hop_attempts++;
      __syncthreads();
    }

    // choose best hop and write back to main n and v_local
    __syncthreads();
    DEBUG_RUN(
      if (t_id == 0) {
        print1DArrayFloat("E_sys_hop after", E_sys_hop, blockDim.x);
      }
    )
    // find out which thread has the best energy solution
    if (t_id == 0) {
      best_t_id = 0;
      E_best = E_sys_hop[0];
      for (int i = 1; i < blockDim.x; i++) {
        if (E_sys_hop[i] < E_best) {
          best_t_id = i;
          E_best = E_sys_hop[i];
        }
      }
      DEBUG_RUN(
        printf("Best hopping t_id=%d\n", best_t_id);
      )
    }
    __syncthreads();
    // take the best solution from the best performing thread
    if (t_id == best_t_id) {
      *E_sys = E_sys_hop[best_t_id];
      for (int i = 0; i < n_dbs; i++) {
        v_local[i] = v_local_l[i];
        n[i] = n_l[i];
      }
      DEBUG_RUN(
        print1DArrayFloat("Best hopped config", n, n_dbs);
      )
    }
    __syncthreads();

    // Annealing schedule update
    if (t_id == 0) {
      // TODO: update best energy & config tracking
      // TODO: every now and then, recalculate system energy from scratch to reset FP errors
      // annealing schedule parameter update
      cycle++;
      *kT = kT_min + (*kT - kT_min) * alpha;
      if (*v_freeze < v_freeze_thresh) {
        *v_freeze += v_freeze_step;
      }
      DEBUG_RUN(printf("new v_freeze=%f\n", *v_freeze);)
      DEBUG_RUN(printf("\n\n");)
    }
    __syncthreads();
  }

  if (t_id == 0) {
    printf("***** End SimAnneal Algorithm Stream %d *****\n", stream_id);
  }
  DEBUG_RUN(
    if (t_id == 0) {
      printf("Final configuration:\n");
      print1DArrayFloat("n", n, n_dbs);
      printf("Final energy: %.3e\n", *E_sys);
    }
  )
  // write-out the final charge configuration found in this stream
  for (int i = t_id; i < n_dbs; i += stride) {
    n_out[i] = n[i];
  }

  DEBUG_RUN(
    if (t_id == 0) {
      printf("Write-out complete.\n");
    }
  )

  // clean up
  // free shared memory
  __syncthreads();
  if (t_id == 0) {
    free(E_sys_hop);
    free(E_delta_hop);
    free(v_ij_s);
    free(v_ext_s);
    free(n);
    free(dn);
    free(pop_changed);
    free(E_sys);
    free(E_delta);
    free(v_local);
    free(v_freeze);
    free(kT);
    free(temp_scalar);
    free(temp_scalar_1);
    free(temp_vec_ndbs_0);
    free(temp_vec_ndbs_1);
  }
  // free local memory
  free(dbm_occ);
  free(db0_occ);
  free(dbp_occ);
  free(v_local_l);
  free(n_l);
  free(from_state);
  free(from_occ_ind);
  free(to_occ_ind);
  free(from_db_ind);
  free(to_db_ind);
  free(accept_hop);
}

__global__ void initVij(int n_dbs, float eps_r, float debye_length, float *db_locs)
{
  int t_id = threadIdx.x + (blockIdx.x * blockDim.x);
  int stride = blockDim.x * gridDim.x;
  
  float Kc = 1./(4. * constants::PI * eps_r * constants::EPS0);
  printf("Initializing v_ij\n");
  printf("t_id=%d, stride=%d, n_dbs=%d\n", t_id, stride, n_dbs);
  for (int i=t_id; i<n_dbs; i+=stride) {
    for (int j=i; j<n_dbs; j++) {
      if (i==j) {
        v_ij[IDX2C(i,j,n_dbs)] = 0;
        continue;
      }
      float r = sqrtf( powf(fabsf(db_locs[IDX2C(i,0,n_dbs)] - db_locs[IDX2C(j,0,n_dbs)]),2) 
                      +powf(fabsf(db_locs[IDX2C(i,1,n_dbs)] - db_locs[IDX2C(j,1,n_dbs)]),2) );
      r *= powf(10, -10); // convert angstrom to m
      v_ij[IDX2C(i,j,n_dbs)] = constants::Q0 * Kc * expf(-r/(debye_length*1e-9)) / r;
      v_ij[IDX2C(j,i,n_dbs)] = v_ij[IDX2C(i,j,n_dbs)];
      DEBUG_RUN(
        printf("r(%d,%d)=%.3e, v_ij[%d,%d]=%.3e\n", i, j, r, i, j, v_ij[IDX2C(i,j,n_dbs)]);
      )
    }
  }
}

/**
 * Initialize device variables
 */
__global__ void initDeviceVars(int t_n_dbs, float t_muzm, float t_mupz, float t_alpha, float kT_start_in,
                               float kT_min_in, float t_v_freeze_thresh, float v_freeze_step_in, 
                               int t_anneal_cycles, int t_hop_attempt_factor,
                               float *t_v_ext)
{
  // assign simple variables
  printf("Assigning variables\n");
  n_dbs = t_n_dbs;
  muzm = t_muzm;
  mupz = t_mupz;
  alpha = t_alpha;
  kT_start = kT_start_in;
  kT_min = kT_min_in;
  v_freeze_thresh = t_v_freeze_thresh;
  v_freeze_step = v_freeze_step_in;
  anneal_cycles = t_anneal_cycles;
  hop_attempt_factor = t_hop_attempt_factor;

  //hipMalloc(&v_ext, n_dbs*sizeof(float));
  v_ext = new float[n_dbs];
  for (int i=0; i<n_dbs; i++) {
    v_ext[i] = t_v_ext[i];
  }

  //// calculate v_ij using db_locs
  //printf("Initializing v_ij\n");
  //hipMalloc(&v_ij, n_dbs*n_dbs*sizeof(float));
  v_ij = new float[n_dbs*n_dbs];

  //__syncthreads();

  //printf("Device vars initialized\n");
}

__global__ void cleanUpDeviceVars(int num_streams)
{
  //for (int i=0; i<num_streams; i++) {
  //  cublasCheckErrors(hipblasDestroy(cb_hdl[i]));
  //}
  free(v_ij);
  free(v_ext);
}